#include "hip/hip_runtime.h"
#include "gpugameoflife.h"
#include <algorithm>
#include <hip/hip_runtime.h>
#include <>

// TODO: remade with type-alias or with enum;

static int ceilDivision(int value, int divider);

__device__ int getCyclicNumber(int value, int max_value)
{
    /*return (value + max_value) % max_value;*/
    if (value >= max_value) {
        return value % max_value;
    }
    if (value < 0) {
        return max_value + value;
    }
    return value;
}

__device__ char getItem(const char* buffer, int width, int i, int j)
{
    return buffer[i * width + j];
}

__device__ void setItem(char* buffer, int width, int i, int j, char value)
{
    buffer[i * width + j] = value;
}

__device__ char getItemOnCyclicGrid(const char* grid, int width, int height, int i, int j)
{
    return getItem(grid, width,
                   getCyclicNumber(i, height),
                   getCyclicNumber(j, width));
}

__device__ int countLivingNeigbours(const char* grid, int width, int height, int i, int j)
{
    // TODO: with predefined pairs array;
    const int min_add = -1;
    const int max_add = 1;
    int count = 0;
    for (int i_add = min_add; i_add <= max_add; ++i_add) {
        int neighbor_i = getCyclicNumber(i + i_add, height);
        for (int j_add = min_add; j_add <= max_add; ++j_add) {
            if (i_add == 0 && j_add == 0) {
                continue;
            }
            int neighbor_j = getCyclicNumber(j + j_add, width);
            if (getItem(grid, width, neighbor_i, neighbor_j) == LIVING_CELL) {
                ++count;
            }
        }
    }
    return count;
}

__global__ void doLifeStep(const char* input, int width, int height, char* output)
{
    auto i = blockIdx.y * blockDim.y + threadIdx.y;
    auto j = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= height || j >= width) {
        return;
    }
    int count = countLivingNeigbours(input, width, height, i, j);
    char cell = getItem(input, width, i, j);
    if (cell == LIVING_CELL) {
        if (count < 2 || count > 3) {
            setItem(output, width, i, j, DEAD_CELL);
        } else {
            setItem(output, width, i, j, cell);
        }
    } else {
        if (count == 3) {
            setItem(output, width, i, j, LIVING_CELL);
        } else {
            setItem(output, width, i, j, cell);
        }
    }
}

GpuGameHandler::GpuGameHandler(int width, int height)
    : _width(width), _height(height)
{
    auto items_count = width * height;
    hipMalloc(&(this->_input), items_count * sizeof(char));
    hipMalloc(&(this->_output), items_count * sizeof(char));
    this->_block_size = std::min({ MAX_BLOCK_SIZE, width, height });
    this->_grid_height = ceilDivision(height, this->_block_size);
    this->_grid_width = ceilDivision(width, this->_block_size);
}

GpuGameHandler::~GpuGameHandler()
{
    hipFree(this->_input);
    hipFree(this->_output);
}

void GpuGameHandler::doLifeStep(const std::vector<char>& input, std::vector<char>& output)
{
    auto code = hipMemcpy(this->_input, input.data(), input.size() * sizeof(char), hipMemcpyHostToDevice);
    ::doLifeStep<<<dim3(this->_grid_width, this->_grid_height), dim3(this->_block_size, this->_block_size)>>>
        (this->_input, this->_width, this->_height, this->_output);
    code = hipDeviceSynchronize();
    code = hipMemcpy(output.data(), this->_output, output.size() * sizeof(char), hipMemcpyDeviceToHost);
}

int ceilDivision(int value, int divider)
{
    return int(ceil(double(value) / divider));
}
